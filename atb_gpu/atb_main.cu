
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#define threshold 0.0000001

#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4


void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void atb(const double *A, const double *B, double *C, int Ni, int Nj, int Nk);

int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k,Ni,Nj,Nk;

  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);
  h_A = (double *) malloc(sizeof(double)*Ni*Nk);
  h_B = (double *) malloc(sizeof(double)*Nk*Nj);
  h_C = (double *) malloc(sizeof(double)*Ni*Nj);
  h_Cref = (double *) malloc(sizeof(double)*Ni*Nj);
  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = k*Ni+i-1;
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = k*Nj+j+1;
  for (i=0; i<Ni; i++)
   for (j=0; j<Nj; j++) {
    h_C[i*Nj+j] = 0;
    h_Cref[i*Nj+j] = 0;}

  for (i=0;i<Ni;i++)
   for (k=0;k<Nk;k++)
    for (j=0;j<Nj;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[k][j];
     h_Cref[i*Nj+j] += h_A[i+Ni*k]*h_B[k*Nj+j];
  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Nk*Ni*sizeof(double));
  hipMalloc(&d_B, Nj*Nk*sizeof(double));
  hipMalloc(&d_C, Ni*Nj*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Nk*Ni*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nj*Nk*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  int block_size = 32;
  dim3 block;
  block.x = block_size;
  block.y = block_size;
  dim3 grid;
  grid.x = (Ni + block.x - 1) / block.x;
  grid.y = (Nj + block.y - 1) / block.y;
  printf("bloack size x %d \n", block.x);
  printf("block size y %d \n", block.y);
  printf("grid size x %d \n", grid.x);
  printf("grid size y %d \n", grid.y);
  hipEventCreate(&start);
  hipEventCreate(&stop);
  for(int trial=0;trial<5;trial++)
  {
   hipEventRecord(start);
   // Launch kernel
   std::cout << "Ni: " << Ni << std::endl;
   std::cout << "Nj: " << Nj << std::endl;
   std::cout << "Nk: " << Nk << std::endl;
   atb<<<grid, block>>>(d_A, d_B, d_C, Ni,Nj,Nk);
   hipEventRecord(stop);
   checkCUDAError("kernel launch");
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
//   cudaDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, Ni*Nj*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int l = 0; l < Ni*Nj; l++) if (fabs((h_C[l] - h_Cref[l])/h_Cref[l])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", l, h_C[l], h_Cref[l]); return -1;}
   printf("<Ni=%d,Nj=%d,Nk=%d>: Trial %d: GFLOPS: %.2f\n",Ni,Nj,Nk,trial,2.0e-6*Ni*Nj*Nk/elapsedTime);
  }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


