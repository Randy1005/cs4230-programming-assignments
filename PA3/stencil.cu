// For compiling, use "nvcc -O3"; upon logging on to a CHPC node, "module load cuda" is needed to load "nvcc"


#include <hip/hip_runtime.h>
#include <stdio.h>

void checkCUDAError(const char *msg);

#include <stdio.h>

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4097*257)
// #define NUM_ELEMENTS 4096
#define FIXME1 32
#define FIXME2 32

// The FIXMEs indicate where code must replace the FIXMEs.
// The number of output elements is N, out[0:N-1]
// The number of input elements is N+2*RADIUS, IN[0:N+2*RADIUS-1]
// Each element of out holds the sum of a set of 2*RADIUS+1 contiguous elements from in
// The sum of contents in in[0:2*RADIUS] is placed in out[0], 
// sum of elements in in[1:2*RADIUS+1] is placed in out[1], etc.

__global__ void stencil_1d(int *in, int *out, int N) 
{
	__shared__ int tmp[512 + 2 * RADIUS];

	// g : linearized thread index across all threads
	int g = blockDim.x * blockIdx.x + threadIdx.x;
	
	// l : shared memory index
	int l = threadIdx.x + RADIUS;

	// read input into shared memory
	if (g < N) {
		if (threadIdx.x == 0) {
			tmp[l - RADIUS] = in[g];
			tmp[l - RADIUS + 1] = in[g + 1];
			tmp[l - RADIUS + 2] = in[g + 2];
		}

		if (threadIdx.x == 511 || g == N - 1) {
			tmp[l + 1] = in[g + RADIUS + 1];
			tmp[l + 2] = in[g + RADIUS + 2];
			tmp[l + 3] = in[g + RADIUS + 3];
		}
		
		tmp[l] = in[g + RADIUS];
	}

	__syncthreads();

	// calculate stencil
	int sum = 0;
	for (int r = -RADIUS; r <= RADIUS; r++) {
		sum += tmp[l + r];
	}

	out[g] = sum;
}

int main()
{
  int i,r;
  int *d_in, *d_out;
	int *h_in, *h_out, *h_ref;

	h_in = (int*)malloc(sizeof(int) * (NUM_ELEMENTS + 2 * RADIUS));
	h_out = (int*)malloc(sizeof(int) * NUM_ELEMENTS);
	h_ref = (int*)malloc(sizeof(int) * NUM_ELEMENTS);

  // Initialize host data
  for(i = 0; i < (NUM_ELEMENTS + 2*RADIUS); i++ )
    h_in[i] = i; 
  for(i = 0; i < NUM_ELEMENTS; i++)
    h_ref[i] = 0;

  for(i = 0; i < NUM_ELEMENTS; i++)
   for(r = -RADIUS; r <= RADIUS; r++)
    h_ref[i] += h_in[RADIUS+i+r];

  // Allocate space on the device
  hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int));
  hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int));
  checkCUDAError("hipMalloc");

  // Copy input data to device
	hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy");

  // Fix the FIXME's
  
	int num_blk = (NUM_ELEMENTS + 512 - 1) / 512;
	stencil_1d<<<num_blk, 512>>> (d_in, d_out,NUM_ELEMENTS);
  checkCUDAError("Kernel Launch Error:");

  hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);
  checkCUDAError("hipMalloc");

  for( i = 0; i < NUM_ELEMENTS; ++i )
    if (h_ref[i] != h_out[i])
    {
      printf("ERROR: Mismatch at index %d: expected %d but found %d\n",i,h_ref[i], h_out[i]);
      break;
    }

    if (i== NUM_ELEMENTS) printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

