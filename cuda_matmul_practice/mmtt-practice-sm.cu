
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <chrono>

#define BLOCK_SIZE 32 

// matrix multiplication GPU kernel
__global__ void matmul(int *a, int *b, int *c, int N) {

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  
  // begin index of thread blocks
  // to map to tiles
  // in CUDA, y -> row, x -> col
  int a_begin = N * BLOCK_SIZE * by;
  int b_begin = BLOCK_SIZE * bx;
  int submat_sum = 0;

  // the current index of thread blocks
  // to map to tiles
  int a_idx = a_begin + N * ty + tx;
  int b_idx = b_begin + N * ty + tx;

  // declared tiles in shared memory
  __shared__ int as[BLOCK_SIZE][BLOCK_SIZE]; 
  __shared__ int bs[BLOCK_SIZE][BLOCK_SIZE]; 
 

  if ((tx < N) && (ty < N)) {
    for (int t = 0; t < N; t += BLOCK_SIZE) {
      // assigning a's and b's element to tile
      as[ty][tx] = a[a_idx];
      bs[ty][tx] = b[b_idx];
      
      __syncthreads();
      
      // calculate sum for c's submatrix
      for (int k = 0; k < BLOCK_SIZE; k++) {
        submat_sum += as[ty][k] * bs[k][tx]; 
      }

      __syncthreads();

      // move on to the next tile
      // move a_idx along columns
      // move b_idx along rows
      a_idx += BLOCK_SIZE;
      b_idx += (BLOCK_SIZE * N);
    }  
    
    // copy submatrix sum back to GPU global memory
    // from thread shared memory
      
    // calculate beginning index of c's corresponding tile
    int c_begin = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
      
    // copy submatrix sum to c's corresponding element
    c[c_begin + N * ty + tx] = submat_sum;
  }


}


// initialize matrix with N * N random integers
void init_matrix(int *mat, int size) {
  for (int i = 0; i < size; i++) {
    mat[i] = ::rand() % 100;
  }
}

void checkCUDAError(const std::string& msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        std::cerr << "Cuda error: " 
          << msg << ", " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }
}



int main(int argc, char* argv[]) {
  
  // matrix size = N * N
  int N = 1024;
  int bytes = N * N * sizeof(int);
  
  // input arrays
  int *h_a, *h_b;

  // output array
  int *h_c, *h_cref;

  // device arrays
  int *d_a, *d_b, *d_c;

  // allocate memory on host device
  h_a = new int[N * N];
  h_b = new int[N * N];
  h_c = new int[N * N];
  h_cref = new int[N * N];

  // initialize c to all 0s
  for (int i = 0; i < N * N; i++) {
    h_cref[i] = 0;
    h_c[i] = 0;
  }

  // allocate memory on GPU device
  hipMalloc(&d_a, bytes);  
  hipMalloc(&d_b, bytes);  
  hipMalloc(&d_c, bytes);
  checkCUDAError("cuda malloc failure");

  // initialize input matrices
  init_matrix(h_a, N * N);
  init_matrix(h_b, N * N);

  // copy arrays to GPU device
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
  checkCUDAError("cuda memcpy H2D failure");
  
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(N / block.x, N / block.y);
  
  auto beg_gpu = std::chrono::steady_clock::now(); 

  // launch matrix multiplication kernel
  matmul<<<grid, block>>>(d_a, d_b, d_c, N);
 

  auto end_gpu = std::chrono::steady_clock::now(); 
  size_t gpu_time = std::chrono::duration_cast<std::chrono::nanoseconds>(
    end_gpu - beg_gpu
  ).count();
  
  // sychronize device jobs
  hipDeviceSynchronize();


  // copy result back to host
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  checkCUDAError("cuda memcpy D2H failure");

  auto beg_cpu = std::chrono::steady_clock::now();
  // sequential version for correctness check
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      for (int k = 0; k < N; k++) {
        h_cref[i*N + j] += h_a[i*N + k] * h_b[k*N + j]; 
      }
    }
  }
  auto end_cpu = std::chrono::steady_clock::now();
  size_t cpu_time = std::chrono::duration_cast<std::chrono::nanoseconds>(
    end_cpu - beg_cpu
  ).count();


  // check if GPU result matches with CPU result
  for (int i = 0; i < N * N; i++) {
    if (h_cref[i] != h_c[i]) {
      std::cerr << "result does not match\n";
      std::cerr << "h_cref " << i << " = " << h_cref[i] << "\n";
      std::cerr << "h_c " << i << " = " << h_c[i] << "\n";
      std::exit(EXIT_FAILURE); 
    }
  }


  std::cout << "GPU mat mul runtime = " << gpu_time << " ns\n";
  std::cout << "CPU mat mul runtime = " << cpu_time << " ns\n";
  std::cout << "speedup = " << cpu_time / gpu_time << "\n";


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);




  return 0;
}





