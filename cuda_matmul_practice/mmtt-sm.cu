// For compiling, use "nvcc -O3"; upon logging on to a CHPC node, "module load cuda" is needed to load "nvcc"


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#define threshold 0.0000001
#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4
#define TILE_SIZE 32 

void checkCUDAError(const char *msg);

const int DSIZE = 2048;
hipEvent_t start, stop;
float tstart, elapsedTime;

// matrix multiply kernel: C = A * B
__global__ void mmul(const double *A, const double *B, double *C, int ds) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  __shared__ double as[TILE_SIZE][TILE_SIZE];
  __shared__ double bs[TILE_SIZE][TILE_SIZE];
  double sum = 0;

  int a_begin = TILE_SIZE * by;
  int b_begin = ds * TILE_SIZE * bx;
  int a_idx = a_begin + ds * ty + tx;
  int b_idx = b_begin + ds * ty + tx;

  assert(a_idx < ds * ds); 
  assert(b_idx < ds * ds); 

  if ((tx < ds) && (ty < ds)) {
    for (int kt = 0; kt < ds; kt+=TILE_SIZE) {
      as[ty][tx] = A[a_idx];
      bs[ty][tx] = B[b_idx];
      __syncthreads();

      for (int k = 0; k < TILE_SIZE; k++) {
        sum += as[k][ty] * bs[tx][k];
      }
      __syncthreads();
      a_idx += TILE_SIZE * ds;
      b_idx += TILE_SIZE;
    } 
    
    int c_begin = ds * TILE_SIZE * by + TILE_SIZE * bx;
    C[c_begin + ds * ty + tx] = sum;
  }
}

int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;

  h_A = new double[DSIZE*DSIZE];
  h_B = new double[DSIZE*DSIZE];
  h_C = new double[DSIZE*DSIZE];
  h_Cref = new double[DSIZE*DSIZE];
  for (i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = i-1;
    h_B[i] = i+1;
    h_C[i] = 0;
    h_Cref[i] = 0;}

  for (i=0;i<DSIZE;i++)
   for (k=0;k<DSIZE;k++)
    for (j=0;j<DSIZE;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[j][k];
     h_Cref[i*DSIZE+j] += h_A[k*DSIZE+i]*h_B[j*DSIZE+k];
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(double));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(double));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  dim3 block(1,1);  
  dim3 grid(1,1);
  int Bx, By;
  printf("Matrix size: %d\n", DSIZE);
  while(1)
 {
  printf("Specify TB-size-x,TB-size-y: ");
  scanf("%d %d", &Bx,&By);
  if ((Bx==0) or (By==0)) break;
  block.x = Bx;
  block.y = By;
  grid.x = DSIZE / block.x;
  grid.y = DSIZE / block.y;

  for(int trial=0;trial<5;trial++)
  {
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);
   // Launch kernel
   mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
   checkCUDAError("kernel launch");
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
   hipDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int i = 0; i < DSIZE*DSIZE; i++) if (fabs((h_C[i] - h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
   printf("<BX=%d,BY=%d>: Trial %d: GFLOPS: %.2f\n",Bx,By,trial,2.0e-6*DSIZE*DSIZE*DSIZE/elapsedTime);
  }
 }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

